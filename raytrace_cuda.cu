#include "hip/hip_runtime.h"
/*
Ray-tracing code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source or binary form, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher (idea from Ronald Rahaman)
*/

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "BMP43805351.h"

static const int ThreadsPerBlock = 1024;


static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

static void prepare(int frames, int width, long* ball_y)
{
    const int semiwidth = width / 2;
    ball_y[0] = semiwidth;
    float vel = 0.0f;

    for(int i = 1; i < frames; i++)
    {
        ball_y[i] = ball_y[i-1] + vel;
        vel -= width * 0.0005f;  // acceleration
        vel *= 0.998f; // dampening
        if (ball_y[i] < -semiwidth) {
            ball_y[i] = -width - ball_y[i];
            vel = -vel;
        }
    }
}

static __global__ void raytrace(const int width, const int frames, unsigned char* const pic, long* ball_y)
{

  const long i = threadIdx.x + blockIdx.x * (long)blockDim.x;
  if(i > width * width * frames)
  {
    return;
  }
  const int semiwidth = width / 2;
  const int pix_x = (i % width) - semiwidth;
  const int pix_y = ((i / width) % width) - semiwidth;
  const int frame = i / (width * width);

  // eye is at <0, 0, 0>

  // initialize ball
  const float ball_r = semiwidth / 3;  // radius of ball
  float ball_z = semiwidth * 3;

  // initialize light source
  const float sol_x = semiwidth * -64;
  const float sol_y = semiwidth * 64;
  const float sol_z = semiwidth * -16;

  // compute pixels of each frame
  float ball_x = frame * width * 0.004f - semiwidth;

  // send one ray through each pixel
  const float c = ball_x * ball_x + ball_y[frame] * ball_y[frame] + ball_z * ball_z - ball_r * ball_r;
  const int pix_z = semiwidth * 2;
  const int a = pix_x * pix_x + pix_y * pix_y + pix_z * pix_z;
  const float e = pix_x * ball_x + pix_y * ball_y[frame] + pix_z * ball_z;
  const float d = e * e - a * c;
  if (d >= 0.0f) {  // ray hits ball
    const float ds = sqrtf(d);
    const float k1 = (e + ds) / a;
    const float k2 = (e - ds) / a;
    const float k3 = fminf(k1, k2);
    const float k4 = fmaxf(k1, k2);
    if (k4 > 0.0f) {  // in front of (not behind) eye
      const float k = (k3 > 0.0f) ? k3 : k4;

      // ball surface normal at loc where ray hits
      const float n_x = k * pix_x - ball_x;
      const float n_y = k * pix_y - ball_y[frame];
      const float n_z = k * pix_z - ball_z;

      // vector to light source from point where ray hits
      const float s_x = sol_x - k * pix_x;
      const float s_y = sol_y - k * pix_y;
      const float s_z = sol_z - k * pix_z;

      // cosine between two vectors
      const float p = s_x * n_x + s_y * n_y + s_z * n_z;
      const float ls = sqrtf(s_x * s_x + s_y * s_y + s_z * s_z);
      const float ln = sqrtf(n_x * n_x + n_y * n_y + n_z * n_z);
      const float cos = p / (ls * ln);

      if (cos > 0) {  // is lit by light source
        const unsigned char brightness = cos * 255.0f;
	pic[frame * width * width + (pix_y + semiwidth) * width + (pix_x + semiwidth)] = brightness;
      }
    }
  }
}


int main(int argc, char *argv[])
{
  printf("Ray Tracing v1.0\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "USAGE: %s frame_width number_of_frames\n", argv[0]); exit(-1);}
  int width = atoi(argv[1]);
  if (width < 100) {fprintf(stderr, "ERROR: frame_width must be at least 100\n"); exit(-1);}
  if ((width % 2) != 0) {fprintf(stderr, "ERROR: frame_width must be even\n"); exit(-1);}
  int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "ERROR: number_of_frames must be at least 1\n"); exit(-1);}
  printf("frames: %d\n", frames);
  printf("width: %d\n", width);

  // allocate picture array
  unsigned char* pic = new unsigned char [frames * width * width];
  long* ball_y = new long[frames];
  long* ball_yGPU = new long[frames];
  unsigned char* const picGPU = new unsigned char [frames * width * width];
  if (hipSuccess != hipMalloc((void **)&picGPU, sizeof(unsigned char) * frames * width * width)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
//  CheckCuda();
  if (hipSuccess != hipMalloc((void **)&ball_yGPU, sizeof(long) * frames)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
//  CheckCuda();


  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  prepare(frames, width, ball_y);
  if (hipSuccess != hipMemcpy(ball_yGPU, ball_y, sizeof(long) * frames, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}
// CheckCuda();
 // execute timed code
  raytrace<<<((frames*width*width) + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(width, frames, picGPU, ball_yGPU);

//CheckCuda();
// end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);
  CheckCuda();
  // write result to BMP files
  if ((width <= 256) && (frames <= 80)) {
    for (int frame = 0; frame < frames; frame++) {
      BMP24 bmp(0, 0, width, width);
      for (int y = 0; y < width; y++) {
        for (int x = 0; x < width; x++) {
          bmp.dot(x, y, pic[frame * width * width + y * width + x] * 0x010101);
        }
      }
      char name[32];
      sprintf(name, "raytrace%d.bmp", frame + 1000);
      bmp.save(name);
    }
  }

  // clean up
  hipFree(picGPU);
  hipFree(ball_yGPU);
  delete [] pic;
  return 0;
}
